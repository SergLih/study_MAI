#include "hip/hip_runtime.h"
#include <stdio.h>

#define CSC(call) do { \
	hipError_t res = call;	\
	if (res != hipSuccess) { \
		fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)

texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *dst, int w_in, int h_in, int w_out, int h_out) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x_out, y_out, i, j;
	int cw = w_in / w_out;
	int ch = h_in / h_out;

	for (x_out = idx; x_out < w_out; x_out += offsetx) {
		for (y_out = idy; y_out < h_out; y_out += offsety) {
			int x_in = x_out * cw, y_in = y_out * ch; 
			double sx = 0., sy = 0., sz = 0.;
			int n = 0;
			for(i = 0; i < cw; i++) {
				for(j = 0; j < ch; j++) {
					if(x_in + i >= w_in || y_in + j >= h_in)
						continue;
					n++;
					uchar4 t = tex2D(tex, x_in+i, y_in+j);
					sx += t.x;
					sy += t.y;
					sz += t.z;
				}
			}
			sx /= n;
			sy /= n;
			sz /= n;
			dst[y_out * w_out + x_out] = make_uchar4(sx, sy, sz, 0);
		}
	}
}

int main() {
	int w_in, h_in, w_out, h_out;
	char infilename[255], outfilename[255];
	scanf("%s", infilename);
	scanf("%s", outfilename);
	scanf("%d %d", &w_out, &h_out);
	FILE *in = fopen(infilename, "rb");
	fread(&w_in, sizeof(int), 1 , in);
	fread(&h_in, sizeof(int), 1 , in);


	uchar4 *data = (uchar4*)malloc(sizeof(uchar4) * h_in * w_in);
	fread(data, sizeof(uchar4), h_in * w_in, in);
	fclose(in);

	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	hipMallocArray(&arr, &ch, w_in, h_in);
	hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * h_in * w_in, hipMemcpyHostToDevice);

	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false; 

	hipBindTextureToArray(tex, arr, ch);
	uchar4 *dev_data;
	hipMalloc(&dev_data, sizeof(uchar4) * h_out * w_out);
	kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_data, w_in, h_in, w_out, h_out);
	hipMemcpy(data, dev_data, sizeof(uchar4) * h_out * w_out, hipMemcpyDeviceToHost);

	FILE *out = fopen(outfilename, "wb");
	fwrite(&w_out, sizeof(int), 1, out);
	fwrite(&h_out, sizeof(int), 1, out);
	fwrite(data, sizeof(uchar4), w_out * h_out, out);
	fclose(out);

	hipUnbindTexture(tex);
	hipFreeArray(arr);
	hipFree(dev_data);
	free(data);

	return 0;
}
