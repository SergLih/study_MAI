#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <string>
#include <cstring>
#include "mpi.h"
#include <iomanip>
#include <cmath>
#include <fstream>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/extrema.h>

using namespace std;

#define CSC(call) do { \
	hipError_t pixels = call;	\
	if (pixels != hipSuccess) { \
		fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(pixels)); \
		exit(0); \
	} \
} while (0)


enum BorderDir {
	LeftToRight, RightToLeft,
	UpToDown, DownToUp,
	FrontToBack, BackToFront,
	Out,
};

__host__ __device__ int ind(int i, int j, int k, int wx, int wy) {
	return i + wx * ( j + wy * k );
}


// string get_pid(int pid, int n_proc_x, int n_proc_y)
// {
// 	int idx = pid;
// 	int pz = idx / (n_proc_x * n_proc_y);
//     idx -= (pz * n_proc_x * n_proc_y);
//     int py = idx / n_proc_x;
//     int px = idx % n_proc_x;
//     return "P_" + to_string(pid) + "[" + to_string(px) + " " + to_string(py) + " " + to_string(pz) + "] ";
// }

// string printOutput(int pid, int n_proc_x, int n_proc_y, double *data_h, int bsz_x, int bsz_y, int bsz_z, int iter) {
// 	//ofstream fout(fn, std::ofstream::out);
// 	ostringstream oss;
// 	oss << get_pid(pid, n_proc_y, n_proc_y) + ": " + to_string(iter) << endl;
// 	//fout << fn << endl;
// 	for (int k = 0; k < bsz_z+2; ++k){
// 		for (int j = 0; j < bsz_y+2; ++j) {
// 			for (int i = 0; i < bsz_x+2; ++i) {
// 				oss << scientific << setprecision(6) << data_h[ind(i, j, k, bsz_x+2, bsz_y+2)] << "\t";
// 			}
// 			oss << "\n";
// 		}
// 		oss << "\n==========================\n";
// 	}
// 	return oss.str();
// }

// void print(int bsz_x, int bsz_y, int bsz_z, double *data_h, ofstream &resout) {
// 	for (int k = 1; k <= bsz_z; ++k) {
// 			for(int j = 1; j <= bsz_y; ++j) {
// 				for (int i = 1; i <= bsz_x; ++i) {
// 					resout << scientific << setprecision(6) << data_h[ind(i, j, k, bsz_x+2, bsz_y+2)] << " "; 
// 				}
// 				resout << "\n";
// 			}
// 			resout << "\n";
// 		}
// }

struct cmpAbsDoubles {
	__host__ __device__ bool operator()(double a, double b) {
		return fabs(a) < fabs(b);
	}
};

__global__ void jacobi_kernel(double *data_d, double * new_data_d, 
							  int bsz_x, int bsz_y, int bsz_z, double h_x, double h_y, double h_z) {
	int tidx = threadIdx.x;;
	int tidy = threadIdx.y;
	int tidz = threadIdx.z;
	int ofsx = blockDim.x;
	int ofsy = blockDim.y;
	int ofsz = blockDim.z;
	//for(int k = 0; k <= bsz_z+1; ++k)
	for(int k = tidz; k <= bsz_z + 1; k += ofsz) 
		//for (int j = 0; j <= bsz_y+1; ++j)
		for(int j = tidy; j <= bsz_y + 1; j += ofsy) 
			//for (int i = 0; i <= bsz_x+1; ++i) 
			for(int i = tidx; i <= bsz_x + 1; i += ofsx)
				if(i*j*k == 0 || (bsz_x+1-i)*(bsz_y+1-j)*(bsz_z+1-k)==0) {
					new_data_d[ind(i, j, k, bsz_x+2, bsz_y+2)] = data_d[ind(i, j, k, bsz_x+2, bsz_y+2)];
				} else {
					new_data_d[ind(i, j, k, bsz_x+2, bsz_y+2)] =
																 ((data_d[ind(i+1, j,   k,   bsz_x+2, bsz_y+2)] 
																+  data_d[ind(i-1, j,   k,   bsz_x+2, bsz_y+2)]) *h_x 
																+ (data_d[ind(i,   j+1, k,   bsz_x+2, bsz_y+2)] 
																+  data_d[ind(i,   j-1, k,   bsz_x+2, bsz_y+2)]) *h_y
																+ (data_d[ind(i,   j,   k+1, bsz_x+2, bsz_y+2)] 
																+  data_d[ind(i,   j,   k-1, bsz_x+2, bsz_y+2)]) *h_z)
																/ (2*(h_x + h_y + h_z)); 
				}

	__syncthreads();

	//в data запиcывается разница между новым и старым значением (а в рамке -- нули)
	//for(int k = 0; k <= bsz_z+1; ++k)
	for(int k = tidz; k <= bsz_z + 1; k += ofsz)  
		//for (int j = 0; j <= bsz_y+1; ++j)
		for(int j = tidy; j <= bsz_y + 1; j += ofsy) 
			for(int i = tidx; i <= bsz_x + 1; i += ofsx) 
			//for (int i = 0; i <= bsz_x+1; ++i)
					data_d[ind(i, j, k, bsz_x+2, bsz_y+2)] -= new_data_d[ind(i, j, k, bsz_x+2, bsz_y+2)];
}

int main (int argc, char *argv[]) {
	// Initialize MPI
	MPI::Init(argc,argv);


	//double Time_work = MPI_Wtime();

	// Get the number of processes
	int n_proc_total = MPI::COMM_WORLD.Get_size();
	// Get the ID of the process
	int pid       = MPI::COMM_WORLD.Get_rank();

/*При помощи hipGetDeviceCount(&devCount) можно получить кол-во gpu. 
В зависимости от rank' процесса выбирается одна из видеокарт с помошью
hipSetDevice(rank % devCount). 
Когда на одной машине будет запущено несколько процессов, все видеокарты будут 
более менее равномерно загружены.
*/
	int n_gpus = 0;
	CSC(hipGetDeviceCount(&n_gpus));
	CSC(hipSetDevice(pid % n_gpus));

// На первой строке заданы три числа: размер сетки
// процессов. Гарантируется, что при запуске программы количество процессов будет
// равно произведению этих трех чисел. На второй строке задается размер блока,
// который будет обрабатываться одним процессом: три числа. Далее задается путь к
// выходному файлу, в который необходимо записать конечный результат работы
// программы и точность ε . На последующих строках описывается задача: задаются
// размеры области lx , ly и lz , граничные условия: udown
// , uup , uleft , uright , ufront и uback , и начальное значение u .
 
 	int n_proc_x, n_proc_y, n_proc_z;
 	int bsz_x, bsz_y, bsz_z;
 	string filename;
 	double precision, max_error_iter, max_error_block_h;
 	double lx, ly, lz;
 	double u_down, u_up, u_left, u_right, u_front, u_back;
 	double u_0;

 	if (pid==0) {
 		cin >> n_proc_x >> n_proc_y >> n_proc_z;
 		if(n_proc_x * n_proc_y * n_proc_z != n_proc_total) {
 			cerr << "Incorrect number of processes, should be: -np " << n_proc_x * n_proc_y * n_proc_z << endl;
 			MPI::COMM_WORLD.Abort(1);
 		}
	 	cin >> bsz_x >> bsz_y >> bsz_z;
	 	cin >> filename;
	 	cin >> precision >> lx >> ly >> lz;
	 	cin >> u_down >> u_up >> u_left >> u_right >> u_front >> u_back >> u_0;

	 	cerr << n_proc_x << " " << n_proc_y << " " << n_proc_z << " " << endl;
	 	cerr << bsz_x << " " << bsz_y << " " << bsz_z << endl;
	 	cerr << precision << " " << lx << " " << ly << " " << lz << endl;
	 	cerr << u_down << " " << u_up << " " << u_left << " " << u_right << " " << u_front << " " << u_back << " " << u_0 << endl;

 	} 
	
	MPI::COMM_WORLD.Bcast(&n_proc_x, 1, MPI::INT, 0);
	MPI::COMM_WORLD.Bcast(&n_proc_y, 1, MPI::INT, 0);
	MPI::COMM_WORLD.Bcast(&n_proc_z, 1, MPI::INT, 0);
	MPI::COMM_WORLD.Bcast(&bsz_x, 1, MPI::INT, 0);
	MPI::COMM_WORLD.Bcast(&bsz_y, 1, MPI::INT, 0);
	MPI::COMM_WORLD.Bcast(&bsz_z, 1, MPI::INT, 0);
	MPI::COMM_WORLD.Bcast(&precision, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&lx, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&ly, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&lz, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&u_down, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&u_up, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&u_left, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&u_right, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&u_front, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&u_back, 1, MPI::DOUBLE, 0);
	MPI::COMM_WORLD.Bcast(&u_0, 1, MPI::DOUBLE, 0);
	
	int n_x, n_y, n_z;
	n_x = n_proc_x * bsz_x;
	n_y = n_proc_y * bsz_y;
	n_z = n_proc_z * bsz_z;

	double h_x, h_y, h_z;
	h_x = pow(lx / n_x, -2);
	h_y = pow(ly / n_y, -2);
	h_z = pow(lz / n_z, -2);
	int idx = pid;
	int pz = idx / (n_proc_x * n_proc_y);
    idx -= (pz * n_proc_x * n_proc_y);
    int py = idx / n_proc_x;
    int px = idx % n_proc_x;


    int i_start = px*bsz_x, i_end = (px+1)*bsz_x+1;
    int j_start = py*bsz_y, j_end = (py+1)*bsz_y+1;
    int k_start = pz*bsz_z, k_end = (pz+1)*bsz_z+1;
 //    cout << get_pid(pid, n_proc_x, n_proc_y) 
 //    + "\tx: " + to_string(i_start) + "-" + to_string(i_end) 
	// + "\ty: " + to_string(j_start) + "-" + to_string(j_end) 
	// + "\tz: " + to_string(k_start) + "-" + to_string(k_end) + "\n";
	double *data_h, *new_data_h;
	int n_cells_in_block = (bsz_x+2)*(bsz_y+2)*(bsz_z+2);
	data_h = new double[n_cells_in_block];
	new_data_h = new double[n_cells_in_block];
	for (int i = i_start; i <= i_end; ++i){
		for (int j = j_start; j <= j_end; ++j){
			for (int k = k_start; k <= k_end; ++k){
				int i_local = i - i_start;
				int j_local = j - j_start;
				int k_local = k - k_start;

				int local1d = ind(i_local, j_local, k_local, 
									  bsz_x+2, bsz_y+2);
				//fout << to_string(i) + to_string(j) + to_string(k) + " " + to_string(i_local) + to_string(j_local) + to_string(k_local) + "_" + to_string(local1d) +  "\n" ;

				if(i == 0)
					data_h[local1d] = u_left;
				else if(i == n_x+1)
					data_h[local1d] = u_right;
				if (j == 0)
					data_h[local1d] = u_front;
				else if(j == n_y+1)
					data_h[local1d] = u_back;
				if (k == 0)
					data_h[local1d] = u_down;
				else if(k == n_z+1)
					data_h[local1d] = u_up;
				
				if(i*j*k > 0 && (n_x+1-i)*(n_y+1-j)*(n_z+1-k)>0)
					data_h[local1d] = u_0;
			}
		}
	}


	double *data_d, *new_data_d;//, *max_error_block_d;
	//hipMalloc(&max_error_block_d, sizeof(double));
	CSC(hipMalloc(&data_d, sizeof(double) * n_cells_in_block));
	CSC(hipMalloc(&new_data_d, sizeof(double) * n_cells_in_block));
	CSC(hipMemcpy(data_d, data_h, sizeof(double) * n_cells_in_block, hipMemcpyHostToDevice));

	MPI::COMM_WORLD.Barrier();
	
	// Measure the current time
	double start = MPI::Wtime();


	int iter = 0;
	// fout << "\n-----------------------------\n" + 
	// 		printOutput(pid, n_proc_x, n_proc_y, data_h, bsz_x, bsz_y, bsz_z, iter) + 
	// 		"\n-----------------------------\n";
	//ofstream fout("out_P" + to_string(pid), std::ofstream::out);
	do {

		iter++;
		// if(iter%20 == 0 && pid == 0)
		// 	cerr << "iter " + to_string(iter) + "\n";
		// if(pid==0)
		// 	cerr << "*";
	
		//На первом этапе происходит обмен граничными слоями между процессами



		if(px > 0) {
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "sent LTR to "  
			//       + get_pid(ind(px-1, py, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";
			double * border_send_h = new double[bsz_y*bsz_z];
			for (int j = 1; j <= bsz_y; ++j) 
				for (int k = 1; k <= bsz_z; ++k) 
					border_send_h[(k-1)+(j-1)*bsz_z] = data_h[ind(1, j, k, bsz_x+2, bsz_y+2)];

			MPI::COMM_WORLD.Send(border_send_h,	bsz_y*bsz_z, MPI::DOUBLE, 
				ind(px-1, py, pz, n_proc_x, n_proc_y), BorderDir::LeftToRight);
			delete[] border_send_h;
		}

		if(px < n_proc_x - 1) {
			double * border_recv_h = new double[bsz_y*bsz_z];
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "recv LTR fr "  
			//       + get_pid(ind(px+1, py, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";

			MPI::COMM_WORLD.Recv(border_recv_h,	bsz_y*bsz_z, MPI::DOUBLE, 
				ind(px+1, py, pz, n_proc_x, n_proc_y), BorderDir::LeftToRight);
			for (int j = 1; j <= bsz_y; ++j) 
				for (int k = 1; k <= bsz_z; ++k) 
					data_h[ind(bsz_x+1, j, k, bsz_x+2, bsz_y+2)] = border_recv_h[(k-1)+(j-1)*bsz_z];
			delete[] border_recv_h;
		}

		//MPI::COMM_WORLD.Barrier();

		//RTL
		if (px < n_proc_x - 1) {
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "sent RTL to "  
			//       + get_pid(ind(px+1, py, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";
			double * border_send_h = new double[bsz_y*bsz_z];
			for (int j = 1; j <= bsz_y; ++j) 
				for (int k = 1; k <= bsz_z; ++k) 
					border_send_h[(k-1)+(j-1)*bsz_z] = data_h[ind(bsz_x, j, k, bsz_x+2, bsz_y+2)];

			MPI::COMM_WORLD.Send(border_send_h, bsz_y*bsz_z, MPI::DOUBLE, 
				ind(px+1, py, pz, n_proc_x, n_proc_y), BorderDir::RightToLeft);
			delete[] border_send_h;
		}

		if (px > 0) {
			double * border_recv_h = new double[bsz_y*bsz_z];
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "recv RTL fr "  
			//       + get_pid(ind(px-1, py, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";

			MPI::COMM_WORLD.Recv(border_recv_h, bsz_y*bsz_z, MPI::DOUBLE, 
				ind(px-1, py, pz, n_proc_x, n_proc_y), BorderDir::RightToLeft);
			for (int j = 1; j <= bsz_y; ++j) 
				for (int k = 1; k <= bsz_z; ++k) 
					data_h[ind(0, j, k, bsz_x+2, bsz_y+2)] = border_recv_h[(k-1)+(j-1)*bsz_z];

			delete[] border_recv_h;
		}
		//MPI::COMM_WORLD.Barrier();

// 		//FTB

		if(py > 0) {
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "sent FTB to "  
			//       + get_pid(ind(px, py-1, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";
			double * border_send_h = new double[bsz_x*bsz_z];
			for (int k = 1; k <= bsz_z; ++k) 
				memcpy(border_send_h + (k-1)*bsz_x, data_h + ind(1, 1, k, bsz_x+2, bsz_y+2), sizeof(double)*bsz_x);

			MPI::COMM_WORLD.Send(border_send_h,	bsz_x*bsz_z, MPI::DOUBLE, 
				ind(px, py-1, pz, n_proc_x, n_proc_y), BorderDir::FrontToBack);
			delete[] border_send_h;
		}

		if(py < n_proc_y - 1){
			double * border_recv_h = new double[bsz_x*bsz_z];
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "recv FTB fr "  
			//       + get_pid(ind(px, py+1, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";

			MPI::COMM_WORLD.Recv(border_recv_h,	bsz_x*bsz_z, MPI::DOUBLE, 
				ind(px, py+1, pz, n_proc_x, n_proc_y), BorderDir::FrontToBack);
			for (int k = 1; k <= bsz_z; ++k) 
				memcpy(data_h + ind(1, bsz_y+1, k, bsz_x+2, bsz_y+2),  border_recv_h + (k-1)*bsz_x, sizeof(double)*bsz_x);

			delete[] border_recv_h;
		}

		//MPI::COMM_WORLD.Barrier();

		//BTF
		
		if (py < n_proc_y - 1) {
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "sent BTF to "  
			//       + get_pid(ind(px, py+1, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";
			double * border_send_h = new double[bsz_x*bsz_z];
			for (int k = 1; k <= bsz_z; ++k)
				memcpy(border_send_h + (k-1)*bsz_x, data_h + ind(1, bsz_y, k, bsz_x+2, bsz_y+2), sizeof(double)*bsz_x);

			MPI::COMM_WORLD.Send(border_send_h, bsz_x*bsz_z, MPI::DOUBLE, 
				ind(px, py+1, pz, n_proc_x, n_proc_y), BorderDir::BackToFront);
			delete[] border_send_h;
		}

		if (py > 0) {
			double * border_recv_h = new double[bsz_x*bsz_z];
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "recv BTF fr "  
			//       + get_pid(ind(px, py-1, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";

			MPI::COMM_WORLD.Recv(border_recv_h, bsz_x*bsz_z, MPI::DOUBLE, 
				ind(px, py-1, pz, n_proc_x, n_proc_y), BorderDir::BackToFront);
			for (int k = 1; k <= bsz_z; ++k)
				memcpy(data_h + ind(1, 0, k, bsz_x+2, bsz_y+2), border_recv_h + (k-1)*bsz_x, sizeof(double)*bsz_x);
			delete[] border_recv_h;
		}
		//MPI::COMM_WORLD.Barrier();

// 		//UTD

		if(pz > 0) {
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "sent UTD to "  
			//       + get_pid(ind(px, py, pz-1, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";
			double * border_send_h = new double[bsz_x*bsz_y];
			for (int j = 1; j <= bsz_y; ++j)
				memcpy(border_send_h + (j-1)*bsz_x, data_h + ind(1, j, 1, bsz_x+2, bsz_y+2), sizeof(double)*bsz_x);	

			MPI::COMM_WORLD.Send(border_send_h, bsz_x*bsz_y, MPI::DOUBLE, 
				ind(px, py, pz-1, n_proc_x, n_proc_y), BorderDir::UpToDown);
			delete[] border_send_h;
		}

		if(pz < n_proc_z - 1){
			double * border_recv_h = new double[bsz_x*bsz_y];
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "recv UTD fr "  
			//       + get_pid(ind(px, py, pz+1, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";

			MPI::COMM_WORLD.Recv(border_recv_h, bsz_x*bsz_y, MPI::DOUBLE, 
				ind(px, py, pz+1, n_proc_x, n_proc_y), BorderDir::UpToDown);
			for (int j = 1; j <= bsz_y; ++j)
				memcpy(data_h + ind(1, j, bsz_z+1, bsz_x+2, bsz_y+2), border_recv_h + (j-1)*bsz_x, sizeof(double)*bsz_x);
			delete[] border_recv_h;
		}

		//MPI::COMM_WORLD.Barrier();

		//DTU

		if (pz < n_proc_z - 1) {
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "sent DTU to "  
			//       + get_pid(ind(px, py, pz+1, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";
			double * border_send_h = new double[bsz_x*bsz_y];
			for (int j = 1; j <= bsz_y; ++j)
				memcpy(border_send_h + (j-1)*bsz_x, data_h + ind(1, j, bsz_z, bsz_x+2, bsz_y+2), sizeof(double)*bsz_x);

			MPI::COMM_WORLD.Send(border_send_h, bsz_x*bsz_y, MPI::DOUBLE, 
				ind(px, py, pz+1, n_proc_x, n_proc_y), BorderDir::DownToUp);
			delete [] border_send_h;
		}

		if (pz > 0) {
			double * border_recv_h = new double[bsz_x*bsz_y];
			// cerr << "iter " + to_string(iter) + " | " + get_pid(pid, n_proc_x, n_proc_y) + "recv DTU fr "  
			//       + get_pid(ind(px, py, pz-1, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n";

			MPI::COMM_WORLD.Recv(border_recv_h, bsz_x*bsz_y, MPI::DOUBLE, 
				ind(px, py, pz-1, n_proc_x, n_proc_y), BorderDir::DownToUp);
			for (int j = 1; j <= bsz_y; ++j) 
				memcpy(data_h + ind(1, j, 0, bsz_x+2, bsz_y+2), border_recv_h + (j-1)*bsz_x, sizeof(double)*bsz_x);
			delete[] border_recv_h;
		}
		MPI::COMM_WORLD.Barrier();

//На втором этапе выполняется обновление значений во всех ячейках

		//double * new_data_h = new double[n_cells_in_block];
		//memcpy(new_data_h, data_h, sizeof(double)*n_cells_in_block);
		// fout << "NEW BEFORE\n" + printOutput(pid, n_proc_x, n_proc_y, data_h, bsz_x, bsz_y, bsz_z, iter);
		CSC(hipMemcpy(data_d, data_h, sizeof(double)*n_cells_in_block, hipMemcpyHostToDevice));
		int n_gpu_threads_x = 32, n_gpu_threads_y = 8, n_gpu_threads_z, tmp;

		hipDeviceProp_t prop;
    	hipGetDeviceProperties(&prop, pid % n_gpus);
		tmp = prop.maxThreadsPerBlock;

		//if(!pid)
		//	cerr << tmp << "\n";
		n_gpu_threads_z = tmp / (n_gpu_threads_x * n_gpu_threads_y);
		jacobi_kernel<<<1, dim3(n_gpu_threads_x, n_gpu_threads_y, n_gpu_threads_z)>>>(data_d, new_data_d, bsz_x, bsz_y, bsz_z, h_x, h_y, h_z);
		CSC(hipGetLastError());
		// hipMemcpy(data_h, data_d, sizeof(double)*n_cells_in_block, hipMemcpyDeviceToHost);
		// fout << "DIFF AFTER\n" + printOutput(pid, n_proc_x, n_proc_y, data_h, bsz_x, bsz_y, bsz_z, iter);

		hipMemcpy(data_h, new_data_d, sizeof(double)*n_cells_in_block, hipMemcpyDeviceToHost);
		// fout << "NEW AFTER\n" + printOutput(pid, n_proc_x, n_proc_y, data_h, bsz_x, bsz_y, bsz_z, iter);

		//MPI::COMM_WORLD.Barrier();

		thrust::device_ptr<double> devPtr = thrust::device_pointer_cast(data_d);
		thrust::device_ptr<double> maxPtr = thrust::max_element(devPtr, devPtr + n_cells_in_block, cmpAbsDoubles());
		max_error_block_h = abs(*maxPtr);

		//hipMemcpy(&max_error_block_h, max_error_block_d, sizeof(double), hipMemcpyDeviceToHost);
		// fout << "ERROR BLOCK = " + to_string(max_error_block_h) + "\n";

		//memcpy(data_h, new_data_h, sizeof(double)*n_cells_in_block);
		//delete[] new_data_h;
		
		MPI::COMM_WORLD.Barrier();
		//fout << "data_h AFTER\n" + printOutput(pid, n_proc_x, n_proc_y, data_h, bsz_x, bsz_y, bsz_z, iter);

		// Sum the error of all the processes
		// Output is stored in the variable ’error’ of all processes
		MPI::COMM_WORLD.Allreduce(&max_error_block_h, &max_error_iter, 1, MPI::DOUBLE, MPI::MAX);
		// if(pid==0)
		// 	cerr << "i " + to_string(iter) + ": e=" + to_string(max_error_iter) + "\n";
		// if(iter==10)
		// 	break;

	
	} while(max_error_iter > precision);
	double t = MPI_Wtime() - start;
	if(!pid)
		cerr << "\n" << iter << " " << t << "\n";



	if (pid == 0) {
		ofstream resout(filename, std::ofstream::out);
		for (int pz = 0; pz < n_proc_z; ++pz) {
			for (int k = 0; k < bsz_z; ++k) {
				for (int py = 0; py < n_proc_y; ++py) {
					for (int j = 0; j < bsz_y; ++j) {
						for (int px = 0; px < n_proc_x; ++px) {
							double *temp_data = new double[bsz_x];
							if(px + py + pz == 0) {
								memcpy(temp_data, 
									data_h + ind(1, j+1, k+1, bsz_x+2, bsz_y+2), 
									bsz_x*sizeof(double));
							} else {
								// cerr << " P0 recv row [" + to_string(j) + "," + to_string(k) + "]" +  
								//         + " fr " + get_pid(ind(px, py, pz, n_proc_x, n_proc_y), n_proc_x, n_proc_y) + "\n"; 
								MPI::COMM_WORLD.Recv(temp_data, bsz_x, MPI::DOUBLE, 
													 ind(px, py, pz, n_proc_x, n_proc_y), 
													 j+k*bsz_y);
							}
							for (int i = 0; i < bsz_x; ++i) {
								resout << scientific << setprecision(6) 
								       << temp_data[i] << " ";
							}

							delete[] temp_data;
						}
						resout << "\n";
					}
				}	
				resout << "\n";
			}
		}
		resout.close();
	} else {
		for (int k = 0; k < bsz_z; ++k) {
			for (int j = 0; j < bsz_y; ++j) {
				// cerr << get_pid(pid, n_proc_x, n_proc_y) 
				// + "sent row [" + to_string(j) + "," + to_string(k) + "] to P0\n";  
				MPI::COMM_WORLD.Send(
					data_h + ind(1, j+1, k+1, bsz_x+2, bsz_y+2), 
					bsz_x, MPI::DOUBLE, 0, j+k*bsz_y);
			}
		}
	}

	delete[] data_h;
	delete[] new_data_h;

	CSC(hipFree(data_d));
	CSC(hipFree(new_data_d));
	// Terminate MPI
	//fout.close();
	MPI::Finalize();
}
